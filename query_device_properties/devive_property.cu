
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        printf("Device %d: %s\n", device, deviceProp.name);
        printf("  Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Number of SMs: %d\n", deviceProp.multiProcessorCount);
        printf("\n");
    }

    return 0;
}
