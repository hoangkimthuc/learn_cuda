
#include <hip/hip_runtime.h>
#include <iostream>
__device__ __managed__ int ret[1000];
__global__ void AplusB(int *ret, int a, int b) {
    ret[threadIdx.x] = a + b + threadIdx.x;
}
int main() {
    AplusB<<< 1, 1000 >>>(ret, 10, 100);
    hipDeviceSynchronize();
    for(int i = 0; i < 1000; i++)
        printf("%d: A+B = %d\n", i, ret[i]);
    hipFree(ret);
    return 0;
}