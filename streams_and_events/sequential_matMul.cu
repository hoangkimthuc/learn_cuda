
#include <hip/hip_runtime.h>
#include <stdio.h>
// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 8

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void async_MatMul(const Matrix A, const Matrix B, Matrix C)
{    
    // cudaStream_t stream[2];
    // for (int i = 0; i < 2; ++i)
    //     cudaStreamCreate(&stream[i]);

    // Load A1 and B1 to device memory
    Matrix d_A1;
    d_A1.width = A.width; d_A1.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A1.elements, size);
    hipMemcpyAsync(d_A1.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B1;
    d_B1.width = B.width; d_B1.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B1.elements, size);
    hipMemcpyAsync(d_B1.elements, B.elements, size,
               hipMemcpyHostToDevice);
    
    // Allocate C1 in device memory
    Matrix d_C1;
    d_C1.width = C.width; d_C1.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C1.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A1, d_B1, d_C1);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A1, d_B1, d_C1);   

    // Read C from device memory
    hipMemcpyAsync(C.elements, d_C1.elements, size,
               hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A1.elements);
    hipFree(d_B1.elements);
    hipFree(d_C1.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}

int main()
{
    Matrix A, B, C;
    A.width = 128;
    A.height = 128;
    B.width = 128;
    B.height = 128;
    C.width = 128;
    C.height = 128;
    A.elements = (float*)malloc(A.width * A.height * sizeof(float));
    B.elements = (float*)malloc(B.width * B.height * sizeof(float));
    C.elements = (float*)malloc(C.width * C.height * sizeof(float));
    for (int i = 0; i < A.width * A.height; i++)
    {
        A.elements[i] = 2;
        B.elements[i] = 2;
    }
    async_MatMul(A, B, C);
}
