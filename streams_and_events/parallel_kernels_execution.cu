#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void myKernel(float* data, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        data[tid] *= 2.0f;
    }
}

int main() {
    const int size = 65536;
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i)
        hipStreamCreate(&stream[i]);
    float* hostPtr1;
    float* hostPtr2;

    hipHostMalloc(&hostPtr1, size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&hostPtr2, size * sizeof(float), hipHostMallocDefault);
    float* devicePtr1;
    float* devicePtr2;

    hipMalloc(&devicePtr1, size * sizeof(float));
    hipMalloc(&devicePtr2, size * sizeof(float));
    // Initialize hostPtr
    // for (int i = 0; i < size; ++i){    
    //     hostPtr1[i] = i;
    //     hostPtr2[i] = i;
    //     cout<<"hostPtr1["<<i<<"]"<<hostPtr1[i]<<endl;
    //     cout<<"hostPtr2["<<i<<"]"<<hostPtr2[i]<<endl;
    // }

    // Copy hostPtr to devicePtr asynchronously
    hipMemcpyAsync(devicePtr1, hostPtr1, size * sizeof(float), hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(devicePtr2, hostPtr2, size * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    // Launch kernel asynchronously on stream[1]
    int blockSize = 32;
    int numBlocks = (size + blockSize - 1) / blockSize;
    myKernel<<<numBlocks, blockSize, 0, stream[0]>>>(devicePtr1, size);
    myKernel<<<numBlocks, blockSize, 0, stream[1]>>>(devicePtr2, size);
    // Copy devicePtr back to hostPtr asynchronously
    hipMemcpyAsync(hostPtr1, devicePtr1, size * sizeof(float), hipMemcpyDeviceToHost, stream[0]);
    hipMemcpyAsync(hostPtr2, devicePtr2, size * sizeof(float), hipMemcpyDeviceToHost, stream[1]);
    // Wait for all operations to complete
    hipStreamSynchronize(stream[0]);
    hipStreamSynchronize(stream[1]);
    // Print the data
    // for (int i = 0; i < size; ++i) {
    //     cout << hostPtr1[i] << " ";
    //     cout << hostPtr2[i] << " ";

    // cout << endl;
    // }
    // Free memory and streams
    hipFree(devicePtr1);
    hipHostFree(hostPtr1);
    hipFree(devicePtr2);
    hipHostFree(hostPtr2);
    for (int i = 0; i < 2; ++i)
        hipStreamDestroy(stream[i]);
    return 0;
}