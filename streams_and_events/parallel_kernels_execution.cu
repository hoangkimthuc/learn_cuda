#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void myKernel(float* data, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        data[tid] *= 2.0f;
    }
}

int main() {
    const int size = 1024;
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i)
        hipStreamCreate(&stream[i]);
    float* hostPtr1;
    float* hostPtr2;

    hipHostMalloc(&hostPtr1, size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&hostPtr2, size * sizeof(float), hipHostMallocDefault);
    float* devicePtr1;
    float* devicePtr2;

    hipMalloc(&devicePtr1, size * sizeof(float));
    hipMalloc(&devicePtr2, size * sizeof(float));

    // Copy hostPtr to devicePtr asynchronously
    hipMemcpyAsync(devicePtr1, hostPtr1, size * sizeof(float), hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(devicePtr2, hostPtr2, size * sizeof(float), hipMemcpyHostToDevice, stream[1]);
    // Launch kernel asynchronously on stream[1]
    int blockSize = 32;
    int numBlocks = (size + blockSize - 1) / blockSize;
    myKernel<<<numBlocks, blockSize, 0, stream[0]>>>(devicePtr1, size);
    myKernel<<<numBlocks, blockSize, 0, stream[0]>>>(devicePtr1, size);
    hipMemcpyAsync(hostPtr1, devicePtr1, size * sizeof(float), hipMemcpyDeviceToHost, stream[0]);
    
    myKernel<<<numBlocks, blockSize, 0, stream[1]>>>(devicePtr2, size);
    // Copy devicePtr back to hostPtr asynchronously   
    hipMemcpyAsync(hostPtr2, devicePtr2, size * sizeof(float), hipMemcpyDeviceToHost, stream[1]);
    // Wait for all operations to complete
    hipStreamSynchronize(stream[0]);
    hipStreamSynchronize(stream[1]);
    // Print the data
    // for (int i = 0; i < size; ++i) {
    //     cout << hostPtr1[i] << " ";
    //     cout << hostPtr2[i] << " ";

    // cout << endl;
    // }
    // Free memory and streams
    hipFree(devicePtr1);
    hipHostFree(hostPtr1);
    hipFree(devicePtr2);
    hipHostFree(hostPtr2);
    for (int i = 0; i < 2; ++i)
        hipStreamDestroy(stream[i]);
    return 0;
}