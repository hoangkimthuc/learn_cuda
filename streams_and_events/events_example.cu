#include <iostream>
#include <hip/hip_runtime.h>

#define N 10

__global__ void kernel(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        a[tid] = tid;
        b[tid] = tid * tid;
        c[tid] = 0;
    }
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    kernel<<<(N+255)/256, 256, 0, stream1>>>(d_a, d_b, d_c);
    kernel<<<(N+255)/256, 256, 0, stream2>>>(d_a, d_b, d_c);

    hipEvent_t event;
    hipEventCreate(&event);
    hipEventRecord(event, stream2);
    hipStreamWaitEvent(stream1, event, 0);

    hipMemcpyAsync(a, d_a, size, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(b, d_b, size, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(c, d_c, size, hipMemcpyDeviceToHost, stream1);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " " << b[i] << " " << c[i] << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(event);

    return 0;
}