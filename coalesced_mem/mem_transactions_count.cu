#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32
#define STRIDE 2
#define STEP 0
template <typename T>
__global__ void add(T *a) {
    int tid = (blockIdx.x*blockDim.x + threadIdx.x)*STRIDE + STEP;
    // __shared__ T cache[BLOCK_SIZE];
    // cache[threadIdx.x] = a[tid];
    // a[tid] = cache[threadIdx.x] + 1.0f;
    a[tid] = a[tid] + 1.0f;
}
template <typename T>
void runTest() {
    T *a_h, *a_d;
    
    // allocate memory on host and device
    a_h = (T *)malloc(sizeof(T)*BLOCK_SIZE*STRIDE);
    hipMalloc((void **)&a_d, sizeof(T)*BLOCK_SIZE*STRIDE);
    // initialize host array and copy it to CUDA device
    for (int i = 0; i < BLOCK_SIZE*STRIDE; i++) {
        a_h[i] = (T)i;
    }
    hipMemcpy(a_d, a_h, sizeof(T)*BLOCK_SIZE*STRIDE, hipMemcpyHostToDevice);
    // launch kernel
    add<<<1,BLOCK_SIZE>>>(a_d);
    // copy results back to host
    hipMemcpy(a_h, a_d, sizeof(T)*BLOCK_SIZE*STRIDE, hipMemcpyDeviceToHost);

    // free memory
    free(a_h);
    hipFree(a_d);
}
int main() {
    runTest<float>();
    return 0;
}