#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matmul_kernel(int* A, int* B, int* C, int L, int M, int N) {
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (col < L && row < N) {
        int C_idx = col*N + row;
        for (int k = 0; k < M; k++) {
            int A_idx = col*M + k;
            int B_idx = row*M + k;
            C[C_idx] += A[A_idx] * B[B_idx];
        }
    }
}

void print_matrix(int* matrix, int r, int c) {
    for (int row = 0; row < r; row++) {
        for (int col = 0; col < c; col++) {
            std::cout << matrix[(row*c + col)] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void transpose_matrix(int* matrix, int r, int c) {
    int* temp = new int[r*c];
    for (int row = 0; row < r; row++) {
        for (int col = 0; col < c; col++) {            
            temp[(col*r + row)] = matrix[(row*c + col)];
        }
    }
    for (int i = 0; i < r*c; i++) {
        matrix[i] = temp[i];
    }
    delete[] temp;
}

int main() {
    int L = 4096; // the size of the matrices
    int M = 4096;
    int N = 4096;

    // Allocate memory for matrices A, B, and C on the host
    int* A = new int[L*M];
    int* B = new int[M*N];
    int* C = new int[L*N];

    // Initialize matrice A with some values
    for (int i = 0; i < L; i++) {
        for (int j = 0; j < M; j++) {
            A[i*L + j] = i + j;            
        }
    }
    // int* A = new int[L*M]{0,1,2,3,
    //                     1,2,3,4,
    //                     2,3,4,5,
    //                     3,4,5,6};

    
    // Print input matrix A 
    // std::cout << "Matrix A: " << std::endl;
    // print_matrix(A, L, M);
   
    
    // Initialize matrice B with some values
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            B[i*M + j] = i - j;
        }
    }

    // std::cout << "Matrix B: " << std::endl;
    // print_matrix(B, M, N);
    // Transpose matrix B
    transpose_matrix(B, M, N);
    // int* B = new int[M*N]{0,1,2,3};
    // std::cout << "Matrix B transposed: " << std::endl;
    // print_matrix(B, N, M);
    
    // Allocate memory for matrices A, B, and C on the device
    int* d_A;
    int* d_B;
    int* d_C;

    hipMalloc((void**) &d_A, L*M*sizeof(int));
    hipMalloc((void**) &d_B, M*N*sizeof(int));
    hipMalloc((void**) &d_C, L*N*sizeof(int));


    // Copy matrices A and B to the device
    hipMemcpy(d_A, A, L*M*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, M*N*sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel with 2D grid
    #define BLOCK_SIZE 32
    dim3 numBlocks((L + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    matmul_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, L, M, N);

    // Copy matrix C from the device to the host
    hipMemcpy(C, d_C, L*N*sizeof(int), hipMemcpyDeviceToHost);    

    // Print output matrix C
    // std::cout << "Matrix C: " << std::endl;
    // print_matrix(C, L, N);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}