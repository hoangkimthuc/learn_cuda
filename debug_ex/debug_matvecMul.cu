
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrix_vector_mult(int* A, int* x, int* y, int N, int M) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < M) {
        y[i] += A[i * N + j] * x[j];
    }
}

int main() {
    int N = 4;
    int M = 4;
    int A[N*M], x[M], y[N];
    int *d_A, *d_x, *d_y;
    int size_A = N * M * sizeof(int);
    int size_x = M * sizeof(int);
    int size_y = N * sizeof(int);

    // Initialize input matrices and vectors
    printf("Input Matrix A:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            A[i*N + j] = i + j;
            printf("%d ", A[i*N + j]);
        }
        printf("\n");
    }

    printf("\nInput Vector x:\n");
    for (int j = 0; j < M; j++) {
        x[j] = j;
        printf("%d\n", x[j]);
    }

    // Allocate device memory for input matrices and vectors
    hipMalloc((void **) &d_A, size_A);
    hipMalloc((void **) &d_x, size_x);
    hipMalloc((void **) &d_y, size_y);
    
    // Copy input matrices and vectors from host memory to device memory
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);

    dim3 block_dim(N, M);
    dim3 numBlocks(1,1);
    // Launch kernel with specified block and grid dimensions      
    matrix_vector_mult<<<numBlocks, block_dim>>>(d_A, d_x, d_y, N, M);

    // Copy result from device memory to host memory
    hipMemcpy(y, d_y, size_y, hipMemcpyDeviceToHost);
    
    // Print output vector y
    printf("\nOutput Vector y:\n");
    for (int i = 0; i < N; i++) {
        printf("%d\n", y[i]);
    }

    // Verify the result
    for (int i = 0; i < N; i++) {
        int sum = 0;
        for (int j = 0; j < M; j++) {
            sum += A[i*N + j] * x[j];
        }
        if (y[i] != sum) {
            printf("Error: index %d, expected %d but got %d\n", i, sum, y[i]);
            break;
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}